#include "hip/hip_runtime.h"
//
// Created by wserver on 2020/6/11.
//

#include "compute_disp.h"
#include <algorithm>

__device__ uint8 find_min_128(uint8 *cost) {
  for (int i = 1; i < 4; ++i) {
    cost[threadIdx.x] = min(cost[threadIdx.x], cost[threadIdx.x + i * threadIdx.x]);
  }
  for (int offset = 16; offset >= 1; offset /= 2) {
    cost[threadIdx.x] = min(cost[threadIdx.x], __shfl_down_sync(MASK, cost[threadIdx.x], offset, 32));
  }
  uint8 min = UINT8_MAX;
  if (threadIdx.x == 0)
    min = cost[threadIdx.x];
  return min;
}

__global__ void compute_disp_kernel(uint8 *disparity,
                                    uint32 height, uint32 width,
                                    uint8 *cost) {
  int idx_x = blockIdx.x * 4 + threadIdx.x;
  int idx_y = blockIdx.y * 4 + threadIdx.y;

  if (idx_y < height && idx_x < width) {
    int idx = idx_y * width + idx_x;
    if (idx < height * width) {
      cost[idx] = find_min_128(disparity + idx * 128);
    }
  }
}

ComputeDisparity::ComputeDisparity(uint8 *disparity,
                                   uint32 height, uint32 width,
                                   uint8 *cost)
    : height_(height), width_(width),
      pDisparity_(disparity), pCost_(cost) {}

void ComputeDisparity::inference() {
#if USE_GPU
  compute_disparity_gpu();
#else
  compute_disparity_cpu();
#endif
}

void ComputeDisparity::compute_disparity_gpu() {
  int grid_dim_y = (height_ + 4 - 1) / 4;
  int grid_dim_x = (width_  + 4 - 1) / 4;

  dim3 grid_dim(grid_dim_x, grid_dim_y);
  dim3 block_dim(4, 4);
  compute_disp_kernel<<<grid_dim, block_dim>>>(pDisparity_, height_, width_, pCost_);
}

void ComputeDisparity::compute_disparity_cpu() {
  uint8 *cost_ptr = pCost_;

  for (int32 i = 0; i < height_; ++i) {
    for (int32 j = 0; j < width_; ++j) {
      uint8 min_cost = UINT8_MAX;
      uint8 max_cost = 0;
      int32 best_disparity = 0;

      for (int32 d = 0; d < MAX_DISPARITY; ++d) {
        const uint8 cost = cost_ptr[(i * width_ + j) * MAX_DISPARITY + d];
        if (min_cost > cost) {
          min_cost = cost;
          best_disparity = d;
        }
        max_cost = std::max(max_cost, cost);
      }

      if (max_cost != min_cost) {
        pDisparity_[i * width_ + j] = best_disparity;
      } else {
        pDisparity_[i * width_ + j] = UINT8_MAX;
      }
    }
  }
}