#include "hip/hip_runtime.h"
//
// Created by wserver on 2020/5/31.
//

#include <opencv2/core/core.hpp>
#include "census_transform.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void census_transform_gpu_kernel(
    uint8 *img, uint32 *result,
    int32 img_rows, int32 img_cols,
    int32 w_hf_h_, int32 w_hf_w_) {
  // blockDim.x = blockDim.y = 16
  int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  int tidy = blockDim.y * blockIdx.y + threadIdx.y;

  if ((tidy < img_rows - 2 * w_hf_h_) &&
      (tidx < img_cols - 2 * w_hf_w_)) {
    int center_idx = (tidy + w_hf_h_) * img_cols + tidx + w_hf_w_;
    int result_idx = tidy * (img_cols - 2 * w_hf_h_) + tidx;

    uint32 census_val = 0u;
    for (int32 i = -w_hf_h_; i <= w_hf_h_; ++i) {
      for (int32 j = -w_hf_w_; j <= w_hf_w_; ++j) {
        census_val <<= 1;
        int idx = (tidy + w_hf_h_ + i) * img_cols + (tidx + w_hf_w_ + j);
        if (img[center_idx] > img[idx])
          census_val += 1;
      }
    }

    result[result_idx] = census_val;
  }
}

__global__ void
census_transform_gpu_kernel(const uint8 *im, const uint8 *im2,
                            uint32 *transform, uint32 *transform2,
                            const uint32 rows, const uint32 cols) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int idy = blockIdx.y * blockDim.y + threadIdx.y;

  const int win_cols = (32 + LEFT * 2);   // 32 + 4 * 2 = 40
  const int win_rows = (32 + TOP * 2);    // 32 + 3 * 2 = 38

  __shared__ uint8 window[win_cols * win_rows];
  __shared__ uint8 window2[win_cols * win_rows];

  const int id = threadIdx.y * blockDim.x + threadIdx.x;
  const int sm_row = id / win_cols;
  const int sm_col = id % win_cols;

  const int im_row = blockIdx.y * blockDim.y + sm_row - TOP;
  const int im_col = blockIdx.x * blockDim.x + sm_col - LEFT;
  const bool boundaries = (im_row >= 0 && im_col >= 0 && im_row < rows && im_col < cols);
  window[sm_row * win_cols + sm_col] = boundaries ? im[im_row * cols + im_col] : 0;
  window2[sm_row * win_cols + sm_col] = boundaries ? im2[im_row * cols + im_col] : 0;

  // Not enough threads fill window and window2
  const int block_size = blockDim.x * blockDim.y;
  if (id < (win_cols * win_rows - block_size)) {
    const int id = threadIdx.y * blockDim.x + threadIdx.x + block_size;
    const int sm_row = id / win_cols;
    const int sm_col = id % win_cols;

    const int im_row = blockIdx.y * blockDim.y + sm_row - TOP;
    const int im_col = blockIdx.x * blockDim.x + sm_col - LEFT;
    const bool boundaries = (im_row >= 0 && im_col >= 0 && im_row < rows && im_col < cols);
    window[sm_row * win_cols + sm_col] = boundaries ? im[im_row * cols + im_col] : 0;
    window2[sm_row * win_cols + sm_col] = boundaries ? im2[im_row * cols + im_col] : 0;
  }

  __syncthreads();
  uint32 census = 0;
  uint32 census2 = 0;
  if (idy < rows && idx < cols) {
    for (int k = 0; k < CENSUS_HEIGHT / 2; ++k) {
      for (int m = 0; m < CENSUS_WIDTH; ++m) {
        const uint8 e1 = window[(threadIdx.y + k) * win_cols + threadIdx.x + m];
        const uint8 e2 = window[(threadIdx.y + 2 * TOP - k) * win_cols + threadIdx.x + 2 * LEFT - m];
        const uint8 i1 = window2[(threadIdx.y + k) * win_cols + threadIdx.x + m];
        const uint8 i2 = window2[(threadIdx.y + 2 * TOP - k) * win_cols + threadIdx.x + 2 * LEFT - m];
      }
    }
  }
}

CensusTransform::CensusTransform(int32 img_height, int32 img_width,
                                 int window_height, int window_width,
                                 uint32 *census_l, uint32 *census_r)
    : pCensusL_(census_l), pCensusR_(census_r),
      imgHeight_(img_height), imgWidth_(img_width) {
  w_hf_h_ = window_height / 2;
  w_hf_w_ = window_width / 2;
}

CensusTransform::~CensusTransform() {
}

void CensusTransform::census_inference(void *img_left, void *img_right) {
#if USE_GPU
  census_transform_gpu(reinterpret_cast<uint8 *>(img_left),
                       pCensusL_, imgHeight_, imgWidth_);
  census_transform_gpu(reinterpret_cast<uint8 *>(img_right),
                       pCensusR_, imgHeight_, imgWidth_);
#else
  census_transform_cpu(*reinterpret_cast<cv::Mat *>(img_left), pCensusL_);
  census_transform_cpu(*reinterpret_cast<cv::Mat *>(img_right), pCensusR_);
#endif
}

void CensusTransform::census_transform_cpu(cv::Mat &img,
                                           uint32 *t_result) {
  int img_height = img.rows;
  int img_width = img.cols;

  for (int32 i = w_hf_h_; i < img_height - w_hf_h_; ++i) {
    for (int32 j = w_hf_w_; j < img_width - w_hf_w_; ++j) {
      // central of window
      const uint8 gray_center = img.at<uint8>(i, j);

      uint32 census_val = 0u;
      for (int32 wh = -w_hf_h_; wh <= w_hf_h_; ++wh) {
        for (int32 ww = -w_hf_w_; ww <= w_hf_w_; ++ww) {
          census_val <<= 1;
          const uint8 gray = img.at<uint8>(i + wh, j + ww);
          if (gray_center > gray)
            census_val += 1;
        }
      }

      t_result[(i - w_hf_h_) * (img_width - 2 * w_hf_w_) + (j - w_hf_w_)] = census_val;
    }
  }
}

void CensusTransform::census_transform_gpu(uint8 *img, uint32 *result,
                                           int32 img_rows, int32 img_cols) {
  int32 grid_dim_x = (img_cols - 2 * w_hf_w_ + 16 - 1) / 16;
  int32 grid_dim_y = (img_rows - 2 * w_hf_h_ + 16 - 1) / 16;
  dim3 gradDim(grid_dim_x, grid_dim_y);
  dim3 blockDim(16, 16);
  census_transform_gpu_kernel<<<gradDim, blockDim>>>(img, result, img_rows, img_cols,
                                                     w_hf_h_, w_hf_w_);
}

void CensusTransform::census_transform_gpu(const uint8 *img_l, const uint8 *img_r,
                                           uint32 *transform, uint32 *transform2,
                                           const uint32 rows, const uint32 cols) {

}