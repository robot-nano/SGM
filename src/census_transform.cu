#include "hip/hip_runtime.h"
//
// Created by wserver on 2020/5/31.
//

#include <opencv2/core/core.hpp>
#include "census_transform.h"

CensusTransform::CensusTransform(int window_height, int window_width)
    : window_height_(window_height), window_width_(window_width)
{
  w_hf_h_ = window_height / 2;
  w_hf_w_ = window_width / 2;
}

void CensusTransform::inference(uint32 *l_result, uint32 *r_result,
                                void *img_left, void *img_right) {
  if (USE_GPU) {

  } else {
    census_transform_cpu(*reinterpret_cast<cv::Mat *>(img_left), l_result);
    census_transform_cpu(*reinterpret_cast<cv::Mat *>(img_right), r_result);
  }
}

void CensusTransform::census_transform_cpu(cv::Mat &img,
                                           uint32 *t_result) {
  int img_height  = img.rows;
  int img_width   = img.cols;

  for (int32 i = w_hf_h_; i < img_height - w_hf_h_; ++i) {
    for (int32 j = w_hf_w_; j < img_width - w_hf_w_; ++j) {
      // central of window
      const uint8 gray_center = img.at<uchar>(i, j);

      uint32 census_val = 0u;
      for (int32 wh = -w_hf_h_; wh <= w_hf_h_; ++wh) {
        for (int32 ww = -w_hf_w_; ww <= w_hf_w_; ++ww) {
          census_val <<= 1;
          const uint8 gray = img.at<uchar>(i + wh, j + ww);
          if (gray < gray_center)
            census_val += 1;
        }
      }

      t_result[i * (img_width - 2 * w_hf_w_) + j] = census_val;
    }
  }
}

#if USE_GPU
__global__ void CensusTransform::census_transform_gpu(
    uint8 *img, uint8 *result,
    int32 img_rows, int32 img_cols) {
  // blockDim.x = blockDim.y = 16
  int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  int tidy = blockDim.y * blockDim.y + threadIdx.y;

  if ((tidy < img_cols - 2 * w_hf_h_) &&
      (tidx < img_rows - 2 * w_hf_w_)) {
    int center_idx = (tidy + w_hf_h_) * img_cols + tidx + w_hf_w_;
    int result_idx = tidy * (img_cols - 2 * w_hf_h_) + (tidx - 2 * w_hf_w_);

    uint32 census_val = 0u;
    for (int i = -w_hf_h_; i < w_hf_h_; ++i) {
      for (int j = -w_hf_w_; j < w_hf_w_; ++j) {
        census_val <<= 1;
        int idx = (tidy + w_hf_h_ - i) * img_cols + (tidx + w_hf_w_ - j);
        if (img[center_idx] > img[idx])
          census_val += 1;
      }
    }

    result[result_idx] = census_val;
  }
}
#endif